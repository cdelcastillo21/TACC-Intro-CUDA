#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA kernel to square each element of an array
__global__ void squareArray(int *d_in, int *d_out, int size)
{
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Make sure we don't go out of bounds
    if (tid < size)
    {
        d_out[tid] = d_in[tid] * d_in[tid];
    }
}

int main()
{
    // Array size
    const int N = 10000;
    size_t bytes = N * sizeof(int);
    
    // Host arrays
    int *h_in = (int*)malloc(bytes);
    int *h_out = (int*)malloc(bytes);
    
    // Initialize input array on host
    for (int i = 0; i < N; i++)
    {
        h_in[i] = i;
    }
    
    // Device arrays
    int *d_in, *d_out;
    
    // Allocate memory on the device
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes);
    
    // Copy input array from host to device
    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);
    
    // Set up execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    // Launch the kernel
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    squareArray<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, N);
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    // Copy result back to host
    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);
    
    // Verify results
    for (int i = 0; i < N; i++)
    {
        if (h_out[i] != h_in[i] * h_in[i])
        {
            printf("Verification failed at index %d: expected %d, got %d\n", 
                   i, h_in[i] * h_in[i], h_out[i]);
            break;
        }
    }
    printf("Array squaring completed successfully!\n");
    
    // Print first 10 elements for verification
    printf("First 10 elements of squared array:\n");
    for (int i = 0; i < 10; i++)
    {
        printf("%d² = %d\n", i, h_out[i]);
    }
    
    // Free device memory
    hipFree(d_in);
    hipFree(d_out);
    
    // Free host memory
    free(h_in);
    free(h_out);
    
    return 0;
}
